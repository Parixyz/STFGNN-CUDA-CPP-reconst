#include "hip/hip_runtime.h"
#include "stfgnn_layer.h"
#include <hip/hip_runtime.h>

__global__ void spatial_temporal_convolution(float* input, float* output, float* weights, int input_dim, int output_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < output_dim) {
        float result = 0.0f;
        for (int i = 0; i < input_dim; i++) {
            result += input[i] * weights[i * output_dim + idx];
        }
        output[idx] = result;
    }
}

STFGNNLayer::STFGNNLayer(int input_dim, int output_dim)
    : input_dim(input_dim), output_dim(output_dim) {
    weights.resize(input_dim * output_dim, 0.5f); // Example initialization
}

std::vector<float> STFGNNLayer::forward(const std::vector<float>& input) {
    std::vector<float> output(output_dim);
    float* d_input;
    float* d_output;
    float* d_weights;

    hipMalloc(&d_input, input.size() * sizeof(float));
    hipMalloc(&d_output, output.size() * sizeof(float));
    hipMalloc(&d_weights, weights.size() * sizeof(float));

    hipMemcpy(d_input, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights.data(), weights.size() * sizeof(float), hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int blocks = (output_dim + threads_per_block - 1) / threads_per_block;

    spatial_temporal_convolution<<<blocks, threads_per_block>>>(d_input, d_output, d_weights, input_dim, output_dim);

    hipMemcpy(output.data(), d_output, output.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_weights);

    return output;
}
