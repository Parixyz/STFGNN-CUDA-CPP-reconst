#include "hip/hip_runtime.h"
#include "preprocessing.h"
#include <hip/hip_runtime.h>

__global__ void zscore_normalize(float* data, int size, float mean, float stddev) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = (data[idx] - mean) / stddev;
    }
}

void preprocess_data(std::vector<float>& data, float mean, float stddev) {
    float* d_data;
    hipMalloc(&d_data, data.size() * sizeof(float));
    hipMemcpy(d_data, data.data(), data.size() * sizeof(float), hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int blocks = (data.size() + threads_per_block - 1) / threads_per_block;
    zscore_normalize<<<blocks, threads_per_block>>>(d_data, data.size(), mean, stddev);

    hipMemcpy(data.data(), d_data, data.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_data);
}
